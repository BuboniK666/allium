#include "hip/hip_runtime.h"
// Copyright 2021 Hannah Rittich
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "cuda_vector.hpp"
#include "cuda_util.hpp"
#include <allium/util/assert.hpp>

#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <iterator>
#include <cassert>

namespace allium {

  template <typename T>
  CudaArray<T>::CudaArray(size_t element_count)
    : m_ptr(nullptr)
  {
    hipError_t err;
    if (element_count > 0) {
      err = hipMalloc(&m_ptr, element_count * sizeof(T));
      cuda_check_status(err, "allocate cuda array");
    }
  }

  template <typename T>
  CudaArray<T>::CudaArray(CudaArray&& other)
    : m_ptr(other.m_ptr)
  {
    other.m_ptr = nullptr;
  }

  template <typename T>
  auto CudaArray<T>::operator= (CudaArray&& other) -> CudaArray&
  {
    if (m_ptr != nullptr) {
      hipFree(m_ptr);
    }
    m_ptr = other.m_ptr;
    other.m_ptr = nullptr;

    return *this;
  }

  template <typename T>
  CudaArray<T>::~CudaArray() {
    if (m_ptr != nullptr) {
      hipFree(m_ptr);
    }
  }

  template <typename T>
  void CudaArray<T>::resize(size_t element_count)
  {
    if (m_ptr != nullptr) {
      hipFree(m_ptr);
      m_ptr = nullptr;
    }
    if (element_count > 0) {
      hipMalloc(&m_ptr, element_count * sizeof(T));
    }
  }

  template <typename N, typename ReduceOp, typename MapOp, typename ...Args>
  N cuda_map_reduce(int n, Args ...a)
  {
    using Number = N;
    hipError_t err;

    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count < 1)
      throw std::runtime_error("No CUDA device found");

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    const int grid_size = prop.multiProcessorCount * 2;

    CudaArray<Number> d_partial(grid_size);

    const auto block_size = partial_map_reduce_block_size;
    partial_map_reduce<Number, ReduceOp, MapOp>
                      <<<grid_size, block_size>>>
                      (d_partial.ptr(), n, a...);
    cuda_check_last_status("execute partial_vec_dot");


    CudaArray<Number> d_result(1);
    partial_map_reduce<Number, ReduceOp, cuda_op::Id<N>>
                      <<<1, block_size>>>
                      (d_result.ptr(), grid_size, d_partial.ptr());
    cuda_check_last_status("execute partial_vec_sum");

    Number h_result;
    err = hipMemcpy(&h_result, d_result.ptr(), sizeof(Number), hipMemcpyDeviceToHost);
    cuda_check_status(err, "copy result");

    return h_result;
  }

  template <typename N, typename Op, typename ...Args>
  __global__ void map_kernel(Op op, int n, N* a, Args ...args) {
    int i_thread = threadIdx.x + blockDim.x * blockIdx.x;

    if (i_thread < n) {
      a[i_thread] = op(a[i_thread], args[i_thread]...);
    }
  }

  template <typename N, typename Op, typename ...Args>
  void cuda_map(Op op, int n, N* a, Args ...args)
  {
    const auto block_size = partial_map_reduce_block_size;
    int grid_size = (n + block_size - 1) / block_size;

    map_kernel<<<grid_size, block_size>>>(op, n, a, args...);
    cuda_check_last_status("execute map_kernel");
  }

  

// === CudaVector ============================================================

  template <typename N>
  CudaVector<N>::CudaVector(VectorSpec spec)
    : VectorStorageTrait<CudaVector, N>(spec),
      m_device_data(spec.local_size())
  {
    allium_assert(spec.local_size() == spec.global_size(),
                  "Cuda vectors are not distributed");
  }

  template <typename N>
  CudaVector<N>::CudaVector(const CudaVector& other)
    : VectorStorageTrait<CudaVector, N>(other.spec()),
      m_device_data(other.spec().local_size())
  {
    hipError_t err;
    err = hipMemcpy(m_device_data.ptr(), other.m_device_data.ptr(),
                     other.spec().local_size() * sizeof(Number),
                     hipMemcpyDeviceToDevice);
    cuda_check_status(err, "copy constructor");
  }

  template <typename N>
  CudaVector<N>::~CudaVector() {}

  template <typename N>
  auto CudaVector<N>::operator+=(const CudaVector<N>& rhs) -> CudaVector&
  {
    size_t n = this->spec().local_size();
    cuda_map(cuda_op::Sum<N>(), n, m_device_data.ptr(), rhs.m_device_data.ptr());

    return *this;
  }

  template <typename N>
  auto CudaVector<N>::operator*=(const N& factor) -> CudaVector&
  {
    size_t n = this->spec().local_size();
    cuda_map(cuda_op::MulBy<N>(factor), n, m_device_data.ptr());

    return *this;
  }

  template <typename N>
  void CudaVector<N>::add_scaled(N factor, const CudaVector& other)
  {
    size_t n = this->spec().local_size();

    cuda_map(cuda_op::AddScaled<N>(factor), n, m_device_data.ptr(), other.m_device_data.ptr());
  }

  template <typename N>
  N CudaVector<N>::dot(const CudaVector& rhs) const
  {
    size_t n = this->spec().local_size();

    return cuda_map_reduce<Number, cuda_op::Sum<N>, cuda_op::Prod<N>>(n, m_device_data.ptr(), rhs.m_device_data.ptr());
  }

  template <typename N>
  auto CudaVector<N>::l2_norm() const -> Real
  {
    size_t n = this->spec().local_size();

    return sqrt(cuda_map_reduce<Number, cuda_op::Sum<N>, cuda_op::Square<N>>(n, m_device_data.ptr()));
  }

  template <typename N>
  auto CudaVector<N>::aquire_data_ptr() -> Number*
  {
    hipError_t err;
    size_t n = this->spec().local_size();
    Number* data = new Number[n];

    err = hipMemcpy(data, m_device_data.ptr(),
                     this->spec().local_size() * sizeof(Number),
                     hipMemcpyDeviceToHost);
    cuda_check_status(err, "copy from device to host");

    return data;
  }

  template <typename N>
  void CudaVector<N>::release_data_ptr(Number* data)
  {
    hipError_t err;

    err = hipMemcpy(m_device_data.ptr(), data,
                     this->spec().local_size() * sizeof(Number),
                     hipMemcpyHostToDevice);
    delete [] data;

    cuda_check_status(err, "copy from host to device");
  }

  //ALLIUM_NOEXTERN_N(ALLIUM_CUDA_VECTOR_DECL)
  ALLIUM_CUDA_VECTOR_DECL(,float)
  ALLIUM_CUDA_VECTOR_DECL(,double)
}

